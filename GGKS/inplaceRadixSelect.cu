/* Copyright 2011 Russel Steinbach, Jeffrey Blanchard, Bradley Gordon,
 *   and Toluwaloju Alabi
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/detail/config.h>
#include <inplaceRadix/inplaceRadixSelect_api.h>
#include "keyConversion.cu"

namespace InplaceRadix{


  //Based on stable_radix_sort, in file thrust/detail/device/cuda/detail/stable_radix_sort_merrill.inl
 template<typename RandomAccessIterator, typename T>
 void inplaceRadixSelect(RandomAccessIterator first, RandomAccessIterator last, uint k, T &result)
{
    uint i ;
    typedef typename thrust::iterator_value<RandomAccessIterator>::type K;
    uint num_elements = last - first;
    uint indexOfK = num_elements - k;
    // ensure data is properly aligned
    if (!thrust::detail::util::is_aligned(thrust::raw_pointer_cast(&*first), 2*sizeof(K)))
    {
        thrust::detail::raw_cuda_device_buffer<K> aligned_keys(first, last);
        inplaceRadixSelect(aligned_keys.begin(), aligned_keys.end(), k,result);
        thrust::copy(aligned_keys.begin(), aligned_keys.end(), first);
        return;
    }
    
    InplaceRadix::RadixSortingEnactor<K> sorter(num_elements);
    InplaceRadix::RadixSortStorage<K>    storage(num_elements, indexOfK);
    
    // allocate temporary buffers
    thrust::detail::raw_cuda_device_buffer<int>          temp_spine(sorter.SpineElements());
    thrust::detail::raw_cuda_device_buffer<bool>         temp_from_alt(2);

    // define storage
    storage.d_keys             = thrust::raw_pointer_cast(&*first);
    storage.d_spine            = thrust::raw_pointer_cast(&temp_spine[0]);
    storage.d_from_alt_storage = thrust::raw_pointer_cast(&temp_from_alt[0]);
    hipMalloc(&(storage.d_radix), sizeof(uint));
    // perform the sort

    sorter.EnactSort(storage);

    uint tmp = 0;
    unsigned long long tmpBig = 0;
    //depending on the size of the items in the list either
    //put it into an uint, or an unsigned long long, this is done because you cannot 
    //do bit operations on floating point numbers
    if(sizeof(T) == sizeof(uint)){
      //this retrieves the radixes from storage
      for(i = 0; i < sizeof(T)*2;i++){
        tmp |= (storage.radixes[i] <<(((sizeof(T)*8) - 4)-( 4 * i)));
      }
      memcpy(&result, &tmp, sizeof(uint));
    }
    else{
      unsigned long long tmp2 =0;
      for(i = 0; i < sizeof(double)*2;i++){
        tmp2 = storage.radixes[i];
        tmpBig |= (tmp2 <<(60-( 4 * i)));
      }
      
      memcpy(&result, &tmpBig, sizeof(double));

    }

    // temporary storage automatically freed
} 

	
uint inplaceRadixSelectWrapper(uint *d_vec, uint size, uint k){
  uint result;
  thrust::device_ptr<uint> dev_ptr(d_vec);
  inplaceRadixSelect(dev_ptr, dev_ptr + size,k,result);
  return result;
}

float inplaceRadixSelectWrapper(float *d_vec, uint size, uint k){
  float result;
  preProcess<float> pre;
  thrust::device_ptr<uint> dev_ptrNew((uint *)d_vec);
  thrust::for_each(dev_ptrNew, dev_ptrNew + size,pre);
  inplaceRadixSelect(dev_ptrNew, dev_ptrNew + size,k,result);
  postProcess<float>((uint *) &result);
  return result;
}

double inplaceRadixSelectWrapper(double *d_vec, uint size, uint k){
  double result;
  preProcess<double> pre;
  thrust::device_ptr<unsigned long long> dev_ptrNew((unsigned long long *)d_vec);
  thrust::for_each(dev_ptrNew, dev_ptrNew + size,pre);
  inplaceRadixSelect(dev_ptrNew, dev_ptrNew + size,k,result);
  postProcess<double>((unsigned long long *) &result);
  return result;
}

}
